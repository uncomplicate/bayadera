#include "hip/hip_runtime.h"
extern "C" {

#include "Random123/philox.h"
#include <stdint.h>
    
#ifndef R123_0x1p_23f
#define R123_0x1p_23f 1.1920928955078125E-7f
#endif

// Sampling from the uniform distribution
    inline float u01fpt_oo_32_24(uint32_t i) {
        return (0.5f + (i >> 9)) * R123_0x1p_23f;
    }

    __global__ void sample (const uint32_t n, const REAL* params, const uint32_t seed,
                            float4* x, const uint32_t offset_x) {

        const uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid * 4 < n) {
            // Generate uniform(0,1) floats
            philox4x32_key_t key;
            uint32_t* key_v = key.v;
            key_v[0] = seed;
            key_v[1] = 0xdecafaaa;
            philox4x32_ctr_t cnt;
            uint32_t* cnt_v = cnt.v;
            cnt_v[0] = gid;
            cnt_v[1] = 0xf00dcafe;
            cnt_v[2] = 0xdeadbeef;

            const float lambda = - params[0];
            const float k = params[1];

            float4 result;
            result.x = 0.0f;
            result.y = 0.0f;
            result.z = 0.0f;
            result.w = 0.0f;
            uint32_t* rand_uni;
            for (uint32_t i = 0; i < k; i++) {
                cnt_v[3] = i;
                rand_uni = philox4x32(cnt, key).v;
                result.x += log(u01fpt_oo_32_24(rand_uni[0]));
                result.y += log(u01fpt_oo_32_24(rand_uni[1]));
                result.z += log(u01fpt_oo_32_24(rand_uni[2]));
                result.w += log(u01fpt_oo_32_24(rand_uni[3]));
            }
            result.x /= lambda;
            result.y /= lambda;
            result.z /= lambda;
            result.w /= lambda;

            x[offset_x + gid] = result;
        }
    }
}
