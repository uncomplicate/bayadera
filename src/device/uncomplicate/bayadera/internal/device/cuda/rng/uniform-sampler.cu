#include "hip/hip_runtime.h"
extern "C" {

#include "Random123/philox.h"
#include <stdint.h>
    
#ifndef R123_0x1p_23f
#define R123_0x1p_23f 1.1920928955078125E-7f
#endif
    
// Sampling from the uniform distribution
    inline float u01fpt_oo_32_24(uint32_t i) {
        return (0.5f + (i >> 9)) * R123_0x1p_23f;
    }

    __global__ void sample (const uint32_t n, const REAL* params, const uint32_t seed,
                            float4* x, const uint32_t offset_x) {
        
        const uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid * 4 < n) {
            // Generate uniform(0,1) floats
            philox4x32_key_t key;
            uint32_t* key_v = key.v;
            key_v[0] = seed;
            key_v[1] = 0xdecafaaa;
            philox4x32_ctr_t cnt;
            uint32_t* cnt_v = cnt.v;
            cnt_v[0] = gid;
            cnt_v[1] = 0xf00dcafe;
            cnt_v[2] = 0xdeadbeef;
            cnt_v[3] = 0xbeeff00d;

            const float lower = params[0];
            const float range = params[1] - lower;

            uint32_t* rand_uni = philox4x32(cnt, key).v;
            float4 result;
            result.x = u01fpt_oo_32_24(rand_uni[0]) * range + lower;
            result.y = u01fpt_oo_32_24(rand_uni[1]) * range + lower;
            result.z = u01fpt_oo_32_24(rand_uni[2]) * range + lower;
            result.w = u01fpt_oo_32_24(rand_uni[3]) * range + lower;
            x[offset_x + gid] = result;
            
        }
    }
}
