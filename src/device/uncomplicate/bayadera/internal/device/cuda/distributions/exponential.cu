#include "hip/hip_runtime.h"
extern "C" {

    inline REAL exponential_log_unscaled(const REAL lambda, const REAL x) {
        return - lambda * x;
    }

    inline REAL exponential_log(const REAL lambda, const REAL x) {
        return log(lambda) - lambda * x;
    }

// ============= With params ========================================

    inline REAL exponential_mcmc_logpdf(const uint32_t data_len, const uint32_t params_len, const REAL* params,
                                        const uint32_t dim, const REAL* x) {
        return (0.0f < x[0]) ? exponential_log_unscaled(params[0], x[0]) : nanf("NaN");
    }


    inline REAL exponential_logpdf(const uint32_t data_len, const uint32_t params_len, const REAL* params,
                                   const uint32_t dim, const REAL* x) {
        return (0.0f < x[0]) ? exponential_log_unscaled(params[0], x[0]) + params[1] : nanf("NaN");
    }
}
